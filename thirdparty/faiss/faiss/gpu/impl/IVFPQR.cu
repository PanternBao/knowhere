#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */
#include <hip/hip_runtime.h>
#include <faiss/gpu/GpuResources.h>
#include <faiss/gpu/impl/InterleavedCodes.h>
#include <faiss/gpu/impl/RemapIndices.h>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <thrust/host_vector.h>
#include <faiss/gpu/impl/BroadcastSum.cuh>
#include <faiss/gpu/impl/Distance.cuh>
#include <faiss/gpu/impl/FlatIndex.cuh>
#include <faiss/gpu/impl/IVFAppend.cuh>
#include <faiss/gpu/impl/IVFPQR.cuh>
#include <faiss/gpu/impl/L2Norm.cuh>
#include <faiss/gpu/impl/PQCodeDistances.cuh>
#include <faiss/gpu/impl/PQScanMultiPassNoPrecomputed.cuh>
#include <faiss/gpu/impl/PQScanMultiPassPrecomputed.cuh>
#include <faiss/gpu/impl/VectorResidual.cuh>
#include <faiss/gpu/utils/BlockSelectKernel.cuh>
#include <faiss/gpu/utils/ConversionOperators.cuh>
#include <faiss/gpu/utils/CopyUtils.cuh>
#include <faiss/gpu/utils/DeviceDefs.cuh>
#include <faiss/gpu/utils/HostTensor.cuh>
#include <faiss/gpu/utils/MatrixMult.cuh>
#include <faiss/gpu/utils/NoTypeTensor.cuh>
#include <faiss/gpu/utils/Transpose.cuh>
#include <limits>
#include <string>
#include <type_traits>
#include <unordered_map>
#include "IVFPQR.cuh"

using namespace std;
namespace faiss {
namespace gpu {

IVFPQR::IVFPQR(
        GpuResources* resources,
        faiss::MetricType metric,
        float metricArg,
        FlatIndex* quantizer,
        int numSubQuantizers,
        int bitsPerSubQuantizer,
        bool useFloat16LookupTables,
        bool useMMCodeDistance,
        bool interleavedLayout,
        float* pqCentroidData,
        float* refinePqCentroidData,
        IndicesOptions indicesOptions,
        MemorySpace space,
        std::vector<uint8_t> refineCodes,
        int refineNumSubQuantizers,
        int debug_flag)
        : IVFPQ(resources,
                metric,
                metricArg,
                quantizer,
                numSubQuantizers,
                bitsPerSubQuantizer,
                useFloat16LookupTables,
                useMMCodeDistance,
                interleavedLayout,
                pqCentroidData,
                indicesOptions,
                space),
          refinePQ(
                  resources,
                  metric,
                  metricArg,
                  quantizer, // don't use ！
                  refineNumSubQuantizers,
                  bitsPerSubQuantizer,
                  useFloat16LookupTables,
                  useMMCodeDistance,
                  interleavedLayout,
                  refinePqCentroidData,
                  indicesOptions,
                  space,
                  refineCodes,
                  debug_flag),
          debug_flag(debug_flag) {}

IVFPQR::~IVFPQR() {}

__global__ void refinePQDistance(
        Tensor<float, 2, true> queries,
        Tensor<int, 2, true> listIds,
        Tensor<int, 2, true> listOffsets,
        void** listCodes,
        /// (sub q)(code id)(sub dim)
        Tensor<float, 3, true> pqCentroidsMiddleCode_,
        Tensor<float, 3, true> listCoarseCentroids,
        Tensor<float, 3, true> refineCentroidsMiddleCode_,
        Tensor<Index::idx_t, 2, true> vectorIndices, // nq * k*kFactor
        Tensor<uint8_t, 2, true> refineCodes,
        int debug_flag) {
    int dimsPerSubQuantizer = pqCentroidsMiddleCode_.getSize(2);
    int numSubQuantizers_ = pqCentroidsMiddleCode_.getSize(0);
    int refineDimsPerSubQuantizer = refineCentroidsMiddleCode_.getSize(2);
    int dim = dimsPerSubQuantizer * numSubQuantizers_;
    int nq = listIds.getSize(0);
    int topK = listIds.getSize(1);
    int i = blockIdx.x;
    auto queryData = queries[i];
    int j = blockIdx.y;
    int listId = listIds[i][j];
    int listOffset = listOffsets[i][j];
    auto coarseCentroid = listCoarseCentroids[i][j];

    Index::idx_t id = vectorIndices[i][j];
    if (debug_flag & PRINT_RESIDUAL2_CODE) {
        printf("residual 2-id %ld\n", id);
    }

    if (listId == -1) {
        printf("listid is -1\n");
        return;
    }
    //float data = 0;
    for (int currentDim = threadIdx.x; currentDim < dim;
         currentDim += blockDim.x) {
        int q = currentDim / dimsPerSubQuantizer;
        int l = currentDim % dimsPerSubQuantizer;

        uint8_t codeId = ((
                uint8_t*)listCodes[listId])[listOffset * numSubQuantizers_ + q];

        float residual1 = queryData[currentDim] -
                coarseCentroid[currentDim] -
                pqCentroidsMiddleCode_[q][codeId][l];

        int refineQ = currentDim / refineDimsPerSubQuantizer;
        int refineL = currentDim % refineDimsPerSubQuantizer;
        uint8_t refine_codeId = refineCodes[id][refineQ];
        if (debug_flag & PRINT_RESIDUAL2_CODE) {
            printf("residual 2-code-id %d\n", (int)refine_codeId);
        }
        float residual2 = refineCentroidsMiddleCode_[refineQ][refine_codeId][refineL];
        float tmp = residual1 - residual2;
        coarseCentroid[currentDim]= tmp * tmp;
        //data += tmp * tmp;
    }
    //outCodeDistances[i][j] = data;
}

// todo :__reduce_add_sync
__global__ void rollupDistances(
        Tensor<float, 3, true> distances,
        Tensor<float, 2, true> outCodeDistances,
        int debug_flag) {
    int i = blockIdx.x;
    int topK = distances.getSize(1);
    int dim = distances.getSize(2);
    for (int j = threadIdx.x; j < topK; j += blockDim.x) {
        float data = 0;

        for (int m = 0; m < dim; m++) {
            data += distances[i][j][m];
        }
        if (debug_flag & PRINT_DISTANCE) {
            printf("l2 dis %f\t", data);
            if (j == distances.getSize(1) - 1) {
                printf("\n");
            }
        }
        outCodeDistances[i][j] = data;
    }
}

__global__ void runPQResidualVector1(
        Tensor<float, 3, true> residual1,
        Tensor<float, 2, true> queries,
        Tensor<int, 2, true> listIds,
        Tensor<int, 2, true> listOffsets,
        void** listCodes,
        /// (sub q)(sub dim)(code id)
        Tensor<float, 3, true> pqCentroidsInnermostCode_,
        /// (sub q)(code id)(sub dim)
        Tensor<float, 3, true> pqCentroidsMiddleCode_,
        Tensor<float, 3, true> listCoarseCentroids,
        int debug_flag) {
    int DimsPerSubQuantizer = pqCentroidsMiddleCode_.getSize(2);
    int numSubQuantizers_ = pqCentroidsMiddleCode_.getSize(0);
    int dim = DimsPerSubQuantizer * numSubQuantizers_;
    int nq = listIds.getSize(0);
    int topK = listIds.getSize(1);
    int i = blockIdx.x;
    auto queryData = queries[i];
    int j = blockIdx.y;
    int listId = listIds[i][j];
    int listOffset = listOffsets[i][j];
    auto coarseCentroid = listCoarseCentroids[i][j];
    if (listId == -1) {
        printf("listid is -1\n");
        return;
    }
    for (int currentDim = threadIdx.x; currentDim < dim;
         currentDim += blockDim.x) {
        int q = currentDim / DimsPerSubQuantizer;
        int l = currentDim % DimsPerSubQuantizer;

        uint8_t codeId = ((
                uint8_t*)listCodes[listId])[listOffset * numSubQuantizers_ + q];

        // printf("%d\t", i * 128 + pq_m * DimsPerSubQuantizer
        // +l);
        // int currentDim = q * DimsPerSubQuantizer + l;
        //                    printf("%f,%f,%f\n",
        //                           (float)queryData[currentDim],
        //                           (float)coarseCentroid[currentDim],
        //                           (float)pqCentroidsMiddleCode_[q][codeId][l]);
        residual1[i][j][currentDim] = queryData[currentDim] -
                coarseCentroid[currentDim] -
                pqCentroidsMiddleCode_[q][codeId][l];

    }

    if (debug_flag & PRINT_RESIDUAL1) {
        printf("residual 1\n");
        for (int i = 0; i < nq; i++) {
            for (int j = 0; j < topK; j++) {
                for (int k = 0; k < dim; k++) {
                    printf("%f\t", (float)residual1[i][j][k]);
                }
            }
            printf("\n");
        }
    }
}

__global__ void calculateListId(
        Tensor<int, 2, true> listIds,
        Tensor<int, 2, true> listOffsets,
        Tensor<Index::idx_t, 2, true> tmpOutIndices,
        int debug_flag) {
    int nq = listIds.getSize(0);
    int topK = listIds.getSize(1);
    // printf("topK,%d,%d\n",nq,topK);
    int i = blockIdx.x;
    for (int j = threadIdx.x; j < topK; j += blockDim.x) {
        Index::idx_t sl = tmpOutIndices[i][j];

        int list_no = sl >> 32;
        int list_offset = sl & 0xffffffff;
        if (sl == -1) {
            list_no = list_offset = -1;
            printf("list_no error!");
            asm("trap;");
            return;
        }
        listOffsets[i][j] = list_offset;

        if (debug_flag & PRINT_LIST_NO) {
            printf("list_no %d,ofs %d \t", list_no, list_offset);
        }
        listIds[i][j] = list_no;
        // printf("list_no %d\t",(int) listIds[i][j]);
    }
//    if (debug_flag & PRINT_LIST_NO) {
//        printf("\n");
//    }
}

// todo __launch_bounds__(288, 3)
__global__ void pqCodeDistances(
        Tensor<float, 3, true> residual1,
        Tensor<float, 3, true> residual2,
        Tensor<float, 2, true> outCodeDistances,
        int debug_flag) {
    int i = blockIdx.x;
    int topK = residual2.getSize(1);
    int dim = residual1.getSize(2);
    for (int j = threadIdx.x; j < topK; j += blockDim.x) {
        float data = 0;

        for (int m = 0; m < dim; m++) {
            float tmp = residual1[i][j][m] - residual2[i][j][m];
            data += tmp * tmp;
        }
        if (debug_flag & PRINT_DISTANCE) {
            printf("l2 dis %f\t", data);
            if (j == residual1.getSize(1) - 1) {
                printf("\n");
            }
        }
        outCodeDistances[i][j] = data;
    }
}

__device__ void printArray(
        Tensor<int, 2, true> codeDistances,
        const char* str) {
    printf("%s\n", str);
    for (int i = 0; i < codeDistances.getSize(0); ++i) {
        for (int j = 0; j < codeDistances.getSize(1); ++j) {
            printf("%d\t", (int)codeDistances[i][j]);
        }
        printf("\n");
    }
    printf("\n");
}

__device__ void printArray(
        Tensor<float, 2, true> codeDistances,
        const char* str) {
    printf("%s\n", str);
    for (int i = 0; i < codeDistances.getSize(0); ++i) {
        for (int j = 0; j < codeDistances.getSize(1); ++j) {
            printf("%f\t", (float)codeDistances[i][j]);
        }
        printf("\n");
    }
    printf("\n");
}

__global__ void sortByDistance(
        Tensor<float, 2, true> codeDistances,
        Tensor<Index::idx_t, 2, true> codeIndices) {
    int nq = codeDistances.getSize(0);
    int topK = codeDistances.getSize(1);
    int i = blockIdx.x;
    for (int j = 1; j < topK; j++) {
        float key = codeDistances[i][j];
        Index::idx_t value = codeIndices[i][j];
        int k = j - 1;
        while (k >= 0 && codeDistances[i][k] > key) {
            codeDistances[i][k + 1] = (float)codeDistances[i][k];
            codeIndices[i][k + 1] = (int)codeIndices[i][k];
            k--;
        }
        codeDistances[i][k + 1] = key;
        codeIndices[i][k + 1] = value;
    }

    // printArray(codeDistances, "codeDistances");
}

template <typename T>
__host__ void extractData(
        Tensor<T, 2, true>& from,
        Tensor<T, 2, true>& to,
        hipStream_t stream) {
    int n1 = to.getSize(0);
    int n2 = to.getSize(1);
    for (int i = 0; i < n1; i++) {
        //        for (int j = 0; j < n2; j++) {
        //            T tmp = from[i][j];
        //            to[i][j] = tmp;
        //        }
        //        hipMemcpyAsync(
        //                from[i].data(),
        //                to[i].data(),
        //                n2 * sizeof(T),
        //                hipMemcpyDeviceToDevice,
        //                stream);
        fromDevice(from[i].data(), to[i].data(), n2, stream);
    }
}

template <typename T>
__global__ void extractData2(
        Tensor<T, 2, true> from,
        Tensor<T, 2, true> to,
        hipStream_t stream) {
    int n1 = to.getSize(0);
    int n2 = to.getSize(1);
    int i = blockIdx.x;
    for (int j = threadIdx.x; j < n2; j += blockDim.x) {
        T tmp = from[i][j];
        to[i][j] = tmp;
    }
    //        hipMemcpyAsync(
    //                from[i].data(),
    //                to[i].data(),
    //                n2 * sizeof(T),
    //                hipMemcpyDeviceToDevice,
    //                stream);
    // fromDevice(from[i].data(), to[i].data(), n2 * sizeof(T), stream);
}


template <typename T>
__global__ void extractIndex(
        Tensor<T, 2, true> from,
        Tensor<int, 2, true> fromIndex,
        Tensor<T, 2, true> to,
        hipStream_t stream) {
    int n1 = to.getSize(0);
    int n2 = to.getSize(1);
    int i = blockIdx.x;
    for (int j = threadIdx.x; j < n2; j += blockDim.x) {
        T tmp = from[i][fromIndex[i][j]];
        to[i][j] = tmp;
    }
}

void IVFPQR::setPrecomputedCodes(bool enable) {
    if (enable) {
        IVFPQ::setPrecomputedCodes(enable); // todo:
    } else {
        IVFPQ::setPrecomputedCodes(enable);
    }
}

void IVFPQR::query(
        Tensor<float, 2, true>& queries,
        Tensor<uint8_t, 1, true>& bitset,
        int nprobe,
        int topK,
        Tensor<float, 2, true>& outDistances,
        Tensor<Index::idx_t, 2, true>& outIndices) {
    // indicesOptions_ = INDICES_IVF;
    if (debug_flag & PRINT_TIME) {
        cout << "use ivfpqr::query" << endl;
    }
    StopWatch sw = StopWatch::start();
    int realK = kFactor * topK;
    // These are caught at a higher level
    FAISS_ASSERT(nprobe <= GPU_MAX_SELECTION_K);
    FAISS_ASSERT(realK <= GPU_MAX_SELECTION_K);
    FAISS_ASSERT(queries.getSize(1) == dim_);
    FAISS_ASSERT(outDistances.getSize(0) == queries.getSize(0));
    FAISS_ASSERT(outIndices.getSize(0) == queries.getSize(0));
    FAISS_ASSERT(indicesOptions_ == INDICES_GPU_ALL);

    auto stream = resources_->getDefaultStreamCurrentDevice();
    int nq = queries.getSize(0);
    nprobe = std::min(nprobe, quantizer_->getSize());

    // FixMe:tmpOutDistances is useless
    DeviceTensor<float, 2, true> tmpOutDistances(
            resources_, makeTempAlloc(AllocType::Other, stream), {nq, realK});
    DeviceTensor<Index::idx_t, 2, true> vectorIndices(
            resources_, makeTempAlloc(AllocType::Other, stream), {nq, realK});
    DeviceTensor<Index::idx_t, 2, true> listNoAndOffsets(
            resources_, makeTempAlloc(AllocType::Other, stream), {nq, realK});


    // Reserve space for the closest coarse centroids
    // 2 代表二维数组，分配大小queries.getSize(0) * nprobe。
    //第一个模板参数是分配的类型，第二是维度
    // 具体见Tensor-inl.cuh和DeviceTensor-inl.cuh， 分配位置见AllocType
    DeviceTensor<float, 2, true> coarseDistances(
            resources_,
            makeTempAlloc(AllocType::Other, stream),
            {nq, nprobe});
    DeviceTensor<int, 2, true> coarseIndices(
            resources_,
            makeTempAlloc(AllocType::Other, stream),
            {nq, nprobe});


    DeviceTensor<int, 2, true> listIds(
            resources_, makeTempAlloc(AllocType::Other, stream), {nq, realK});
    DeviceTensor<int, 2, true> listOffsets(
            resources_, makeTempAlloc(AllocType::Other, stream), {nq, realK});

    DeviceTensor<float, 3, true> listCoarseCentroids(
            resources_,
            makeTempAlloc(AllocType::Other, stream),
            {nq, realK, dim_});


    DeviceTensor<float, 2, true> codeDistances(
            resources_, makeTempAlloc(AllocType::Other, stream), {nq, realK});

    DeviceTensor<int, 2, true> reRankIndices(
            resources_, makeTempAlloc(AllocType::Other, stream), {nq, topK});

    DeviceTensor<uint8_t, 1, true> coarseBitset(
            resources_, makeTempAlloc(AllocType::Other, stream), {0});

    //一级索引查询
    // Find the `nprobe` closest coarse centroids; we can use int
    // indices both internally and externally
    quantizer_->query(
            queries,
            coarseBitset,
            nprobe,
            metric_,
            metricArg_,
            coarseDistances,
            coarseIndices,
            true);

    if (precomputedCodes_) {
        if (debug_flag & PRINT_TIME) {
            printf("use precomputedCodes_\n");
        }
        FAISS_ASSERT(metric_ == MetricType::METRIC_L2);

        runPQPrecomputedCodes_(
                queries,
                bitset,
                coarseDistances,
                coarseIndices,
                realK,
                tmpOutDistances,
                vectorIndices,
                listNoAndOffsets);
    } else {
        runPQNoPrecomputedCodes_(
                queries,
                bitset,
                coarseDistances,
                coarseIndices,
                realK,
                tmpOutDistances,
                vectorIndices,
                listNoAndOffsets);
    }
    if (debug_flag & PRINT_TIME) {
        hipStreamSynchronize(stream);
        hipDeviceSynchronize();
        sw.stop();
        cout << "ivfpq::query done " << sw.getElapsedTime() << endl;
        sw.restart();
    }

    {
        auto grid = dim3(nq);
        auto block = dim3(min(256, realK));
        calculateListId<<<grid, block, 0, stream>>>(
                listIds, listOffsets, listNoAndOffsets, debug_flag);
    }
    if (debug_flag & PRINT_TIME) {
        hipStreamSynchronize(stream);
        hipDeviceSynchronize();
        sw.stop();
        cout << "calculateListId done " << sw.getElapsedTime() << endl;
        sw.restart();
    }

    //计算query和"result所在的粗聚类"的残差
    quantizer_->reconstruct(listIds, listCoarseCentroids);
    if (debug_flag & PRINT_TIME) {
        hipStreamSynchronize(stream);
        hipDeviceSynchronize();
        sw.stop();
        cout << "reconstruct done " << sw.getElapsedTime() << endl;
        sw.restart();
    }
    //listCoarseCentroids will be computed and rewrite
    {
        auto grid = dim3(nq, realK);
        auto block = dim3(min(dim_, 256));
        refinePQDistance<<<grid, block, 0, stream>>>(
                queries,
                listIds,
                listOffsets,
                deviceListDataPointers_.data().get(),
                pqCentroidsMiddleCode_,
                listCoarseCentroids,
                refinePQ.getPQCentroids(),
                vectorIndices,
                refinePQ.refineCodes_,
                debug_flag);
    }
    {
        auto grid = dim3(nq);
        auto block = dim3(min(256, realK));
        rollupDistances<<<grid, block, 0, stream>>>(
                listCoarseCentroids, codeDistances, debug_flag);
    }
    //    //残差
//    DeviceTensor<float, 3, true> residual1(
//            resources_,
//            makeTempAlloc(AllocType::Other, stream),
//            {nq, realK, dim_});

//    {
//        auto grid = dim3(nq, realK);
//        auto block = dim3(min(dim_, 256));
//        runPQResidualVector1<<<grid, block, 0, stream>>>(
//                residual1,
//                queries,
//                listIds,
//                listOffsets,
//                deviceListDataPointers_.data().get(),
//                pqCentroidsInnermostCode_,
//                pqCentroidsMiddleCode_,
//                listCoarseCentroids,
//                debug_flag);
//    }
//    if (debug_flag & PRINT_TIME) {
//        hipStreamSynchronize(stream);
//        hipDeviceSynchronize();
//        sw.stop();
//        cout << "runPQResidualVector1 done " << sw.getElapsedTime() << endl;
//        sw.restart();
//    }
//
//    if (debug_flag & PRINT_TIME) {
//        hipStreamSynchronize(stream);
//        hipDeviceSynchronize();
//        sw.stop();
//        cout << "convert to cpu index done " << sw.getElapsedTime() << endl;
//        sw.restart();
//    }
//
//        DeviceTensor<float, 3, true> residual2(
//        resources_,
//        makeTempAlloc(AllocType::Other, stream),
//        {nq, realK, dim_});
//    refinePQ.calculateResidualVector2(vectorIndices, residual2);
//    if (debug_flag & PRINT_TIME) {
//        hipStreamSynchronize(stream);
//        hipDeviceSynchronize();
//        sw.stop();
//        cout << "calculateResidualVector2 done " << sw.getElapsedTime() << endl;
//        sw.restart();
//    }
//
//    {
//        auto grid = dim3(nq);
//        auto block = dim3(min(256, realK));
//        pqCodeDistances<<<grid, block, 0, stream>>>(
//                residual1, residual2, codeDistances, debug_flag);
//    }
//    if (debug_flag & PRINT_TIME) {
//        hipStreamSynchronize(stream);
//        hipDeviceSynchronize();
//        sw.stop();
//        cout << "pqCodeDistances done " << sw.getElapsedTime() << endl;
//        sw.restart();
//    }


    {
        runBlockSelect(
                codeDistances,
                outDistances,
                reRankIndices,
                false,
                topK,
                stream);
    }
    if (debug_flag & PRINT_TIME) {
        hipStreamSynchronize(stream);
        hipDeviceSynchronize();
        sw.stop();
        cout << "runBlockSelect done " << sw.getElapsedTime() << endl;
        sw.restart();
    }

    {
        auto grid = dim3(nq);
        auto block = dim3(min(256, topK));
        extractIndex<<<grid, block, 0, stream>>>(
                vectorIndices, reRankIndices, outIndices, stream);
    }
    if (debug_flag & PRINT_TIME) {
        hipStreamSynchronize(stream);
        hipDeviceSynchronize();
        sw.stop();
        cout << "extractIndex done " << sw.getElapsedTime() << endl;
        sw.restart();
    }

    // outIndices.copyFrom(tmp_labels.transpose(0, 1)[0], stream);
}

} // namespace gpu
} // namespace faiss

